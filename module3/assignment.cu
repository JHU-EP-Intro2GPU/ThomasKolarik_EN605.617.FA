//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <stdio.h>


__global__ void add(int * a, int * b, int * c)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    c[thread_idx] = a[thread_idx] + b[thread_idx];
}

__global__ void subtract(int * a, int * b, int * c)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    c[thread_idx] = a[thread_idx] - b[thread_idx];
}

__global__ void mult(int * a, int * b, int * c)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    c[thread_idx] = a[thread_idx] * b[thread_idx];
}

__global__ void mod(int * a, int * b, int * c)
{
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    c[thread_idx] = a[thread_idx] % b[thread_idx];
}

void printArray(const int * const arr, const int xSize, const int ySize)
{
    for (size_t i = 0; i < xSize; ++i)
    {
        for(size_t j = 0; j < ySize; ++j)
        {
            std::cout << arr[i * ySize + j] << " ";
        }
        std::cout << '\n';
    }
    
    std::cout << std::flush;
}

int main(int argc, char** argv)
{
    // read command line arguments
    int totalThreads = 256;
    int blockSize = 256;
    
    if (argc >= 2) {
        totalThreads = atoi(argv[1]);
    }
    if (argc >= 3) {
        blockSize = atoi(argv[2]);
    }
    

    int numBlocks = totalThreads/blockSize;

    // validate command line arguments
    if (totalThreads % blockSize != 0) {
        ++numBlocks;
        totalThreads = numBlocks*blockSize;
        
        printf("Warning: Total thread count is not evenly divisible by the block size\n");
        printf("The total number of threads will be rounded up to %d\n", totalThreads);
    }
    
    int a[totalThreads], b[totalThreads], c[totalThreads];
    
    int *gpu_a, *gpu_b, *gpu_c;

    hipMalloc((void**)&gpu_a, totalThreads * sizeof(int));

    hipMalloc((void**)&gpu_b, totalThreads * sizeof(int));

    hipMalloc((void**)&gpu_c, totalThreads * sizeof(int));
    
    // Create a random generate that will generate random numbers from 0 to 4.
    // Use a set seed so output is deterministic
    unsigned seed = 12345;
    std::default_random_engine gen(seed);
    std::uniform_int_distribution<int> dist(0,4);
    
    for (size_t i = 0; i < totalThreads; ++i)
    {
        a[i] = i;
        b[i] = dist(gen);
    }
    
    std::cout << "A:" << std::endl;
    printArray(a, numBlocks, blockSize);
    std::cout << "B:" << std::endl;
    printArray(b, numBlocks, blockSize);
    
    hipMemcpy(gpu_a, a, totalThreads * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(gpu_b, b, totalThreads * sizeof(int), hipMemcpyHostToDevice);
    
    // Add all of the numbers c[i] = a[i] + b[i];
    add<<<numBlocks, blockSize>>>(gpu_a,gpu_b,gpu_c);
    
    hipMemcpy(c, gpu_c, totalThreads*sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout << "Add: " << std::endl;
    printArray(c, numBlocks, blockSize);
    
    // Subtract all of the numbers c[i] = a[i] - b[i];
    subtract<<<numBlocks, blockSize>>>(gpu_a,gpu_b,gpu_c);
    
    hipMemcpy(c, gpu_c, totalThreads*sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Sub: " << std::endl;
    printArray(c, numBlocks, blockSize);
    
    // Multiply all of the numbers c[i] = a[i] * b[i];
    mult<<<numBlocks, blockSize>>>(gpu_a,gpu_b,gpu_c);
    
    hipMemcpy(c, gpu_c, totalThreads*sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Mult: " << std::endl;
    printArray(c, numBlocks, blockSize);
    
    // Mod all of the numbers c[i] = a[i] % b[i];
    mod<<<numBlocks, blockSize>>>(gpu_a,gpu_b,gpu_c);
    
    hipMemcpy(c, gpu_c, totalThreads*sizeof(int), hipMemcpyDeviceToHost);
    
    std::cout << "Mod: " << std::endl;
    printArray(c, numBlocks, blockSize);
    
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);
    
    return 0;
}
