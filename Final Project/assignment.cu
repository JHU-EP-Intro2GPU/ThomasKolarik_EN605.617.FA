#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <random>
#include <chrono>
#include <stdio.h>

#include <npp.h>

#include <ImagesCPU.h>
#include <ImagesNPP.h>
#include <ImageIO.h>

// Counts the number of alive neighbors for a given square
// array: The array with the game world stored within it
// xSize: The size of the game world in the X direction
// ySize: The size of the game world in the Y direction
// xCoord: The X coordinate of the square to check
// yCoord: the Y coordinate of the square to check
// return: The number of squares around the square that are non-zero.
__device__ unsigned int countAliveNeighbors(const unsigned int * array, const unsigned int xSize, const unsigned int ySize, const unsigned int xCoord, const unsigned int yCoord)
{
    unsigned int aliveNeighbors = 0;
    
    // Since it is impossible to store an infinite game world we are going to 
    // have the game world wrap around that is, the coordinate (0,0) is directly
    // next to (xSize,0), and (0,ySize).
    const unsigned int x0 = (xCoord + xSize - 1) % xSize;
    const unsigned int x1 = (xCoord);
    const unsigned int x2 = (xCoord + 1) % xSize;
    
    const unsigned int y0 = (yCoord + ySize - 1) % ySize;
    const unsigned int y1 = (yCoord);
    const unsigned int y2 = (yCoord + 1) % ySize;
    
    // We unravel the obvious set of loops here to have to skip checking the
    // if condition of the neighbor equaling the current square.
    aliveNeighbors += (array[y0 * xSize + x0] != 0);
    aliveNeighbors += (array[y1 * xSize + x0] != 0);
    aliveNeighbors += (array[y2 * xSize + x0] != 0);
    aliveNeighbors += (array[y0 * xSize + x1] != 0);
    aliveNeighbors += (array[y2 * xSize + x1] != 0);
    aliveNeighbors += (array[y0 * xSize + x2] != 0);
    aliveNeighbors += (array[y1 * xSize + x2] != 0);
    aliveNeighbors += (array[y2 * xSize + x2] != 0);
    
    return aliveNeighbors;
}

// Completes a single iteration of the game world for a single cell
// array: The array with the game world stored within it
// xSize: The size of the game world in the X direction
// ySize: The size of the game world in the Y direction
// neighborsToGrow: The number of neighbors required for a cell to grow if previously dead.
// neighborsToDie: The number of neighbors at which the cell will die due to loneliness.
__global__ void progressTime(const unsigned int * array, unsigned int * result, const unsigned int xSize, const unsigned int ySize, const unsigned int neighborsToGrow, const unsigned int neighborsToDie)
{
    const unsigned int xCoord = ((blockIdx.x * blockDim.x) + threadIdx.x) % xSize;
    const unsigned int yCoord = ((blockIdx.x * blockDim.x) + threadIdx.x) / xSize;
    const unsigned int sqIndx = yCoord * xSize + xCoord;
    
    unsigned int aliveNeighbors = countAliveNeighbors(array, xSize, ySize, xCoord, yCoord);
    
    // This line wraps up all of the growing/dying mechanics of the game. In the normal game of life, neighrborsToGrow is 3
    // and neighborsToDie is 1. So the following reduces to aliveNeighbors == neighborsToGrow || (array[xCoord][yCoord] && aliveNeighbors == 2).
    result[sqIndx] = (aliveNeighbors == neighborsToGrow || (array[sqIndx] && aliveNeighbors > neighborsToDie)) && aliveNeighbors <= neighborsToGrow;
}

// Executes the device (gpu) version of the game of life algorithm.
// array: The array with the game world stored within it
// xSize: The size of the array in the X direction
// ySize: The size of the array in the Y direction
// neighborsToGrow: The number of neighbors required for a cell to grow if previously dead.
// neighborsToDie: The number of neighbors at which the cell will die due to loneliness.
void executeDevice(const unsigned int * array, const unsigned int xSize, const unsigned int ySize, const unsigned int neighborsToGrow, const unsigned int neighborsToDie)
{
    auto startTime = std::chrono::system_clock::now();
    unsigned int * result = (unsigned int*)calloc(xSize * ySize, sizeof(unsigned int));
    
    unsigned int * gpu_array;
    unsigned int * gpu_result;
    
    hipMalloc((void**)&gpu_array,  xSize * ySize * sizeof(unsigned int));
    hipMalloc((void**)&gpu_result, xSize * ySize * sizeof(unsigned int));
    
    hipMemcpy(gpu_result, result, xSize * ySize * sizeof(unsigned int), hipMemcpyHostToDevice);
    
    hipFree(gpu_array);
    hipFree(gpu_result);
    
    auto endTime = std::chrono::system_clock::now();
    std::chrono::duration<double> totalTime = endTime-startTime;
    std::cout << "Device execution took: " << totalTime.count() << " seconds." << std::endl;
}

// Prints the given array out to the console
// array: The array with the game world stored within it
// xSize: The size of the array in the X direction
// ySize: The size of the array in the Y direction
// neighborsToGrow: The number of neighbors required for a cell
void printArray(const unsigned int * const array, const unsigned int xSize, const unsigned int ySize)
{
    for (size_t y = 0; y < ySize; ++y)
    {
        for(size_t x = 0; x < xSize; ++x)
        {
            std::cout << array[y * xSize + x] << " ";
        }
        std::cout << '\n';
    }
    
    std::cout << std::flush;
}

// Counts the number of alive neighbors for a given square
// array: The array with the game world stored within it
// xSize: The size of the game world in the X direction
// ySize: The size of the game world in the Y direction
// xCoord: The X coordinate of the square to check
// yCoord: the Y coordinate of the square to check
// return: The number of squares around the square that are non-zero.
unsigned int hostCountAliveNeighbors(const unsigned int * array, const unsigned int xSize, const unsigned int ySize, const unsigned int xCoord, const unsigned int yCoord)
{
    unsigned int aliveNeighbors = 0;
    
    // Since it is impossible to store an infinite game world we are going to 
    // have the game world wrap around that is, the coordinate (0,0) is directly
    // next to (xSize,0), and (0,ySize).
    const unsigned int x0 = (xCoord + xSize - 1) % xSize;
    const unsigned int x1 = (xCoord);
    const unsigned int x2 = (xCoord + 1) % xSize;
    
    const unsigned int y0 = (yCoord + ySize - 1) % ySize;
    const unsigned int y1 = (yCoord);
    const unsigned int y2 = (yCoord + 1) % ySize;
    
    // We unravel the obvious set of loops here to have to skip checking the
    // if condition of the neighbor equaling the current square.
    aliveNeighbors += (array[y0 * xSize + x0] != 0);
    aliveNeighbors += (array[y1 * xSize + x0] != 0);
    aliveNeighbors += (array[y2 * xSize + x0] != 0);
    aliveNeighbors += (array[y0 * xSize + x1] != 0);
    aliveNeighbors += (array[y2 * xSize + x1] != 0);
    aliveNeighbors += (array[y0 * xSize + x2] != 0);
    aliveNeighbors += (array[y1 * xSize + x2] != 0);
    aliveNeighbors += (array[y2 * xSize + x2] != 0);
    
    return aliveNeighbors;
}

// Completes a single iteration of the game world for a single cell
// array: The array with the game world stored within it
// xSize: The size of the game world in the X direction
// ySize: The size of the game world in the Y direction
// neighborsToGrow: The number of neighbors required for a cell to grow if previously dead.
// neighborsToDie: The number of neighbors at which the cell will die due to loneliness.
void hostProgressTime(const unsigned int * array, unsigned int * result, const unsigned int xSize, const unsigned int ySize, const unsigned int neighborsToGrow, const unsigned int neighborsToDie)
{
    for (unsigned int y = 0; y < ySize; ++y)
    {
        for (unsigned int x = 0; x < xSize; ++x)
        {
            const unsigned int sqIndx = y * xSize + x;
            unsigned int aliveNeighbors = hostCountAliveNeighbors(array, xSize, ySize, x, y);
            
            // This line wraps up all of the growing/dying mechanics of the game. In the normal game of life, neighrborsToGrow is 3
            // and neighborsToDie is 1. So the following reduces to aliveNeighbors == neighborsToGrow || (array[xCoord][yCoord] && aliveNeighbors == 2).
            result[sqIndx] = (aliveNeighbors == neighborsToGrow || (array[sqIndx] && aliveNeighbors > neighborsToDie)) && aliveNeighbors <= neighborsToGrow;
        }
    }
}

// Executes the host (cpu) version of the game of life algorithm.
// array: The array with the game world stored within it
// xSize: The size of the array in the X direction
// ySize: The size of the array in the Y direction
// neighborsToGrow: The number of neighbors required for a cell to grow if previously dead.
// neighborsToDie: The number of neighbors at which the cell will die due to loneliness.
void executeHost(const unsigned int * array, const unsigned int xSize, const unsigned int ySize, const unsigned int neighborsToGrow, const unsigned int neighborsToDie)
{
    auto startTime = std::chrono::system_clock::now();
    unsigned int * result = (unsigned int*)calloc(xSize * ySize, sizeof(unsigned int));
    hostProgressTime(array, result, xSize, ySize, neighborsToGrow, neighborsToDie);
    auto endTime = std::chrono::system_clock::now();
    std::chrono::duration<double> totalTime = endTime-startTime;
    std::cout << "Host execution took: " << totalTime.count() << " seconds." << std::endl;
    
    printArray(array, xSize, ySize);
    printArray(result, xSize, ySize);
}

// Takes in a file name and parses it to setup the initial game state.
// argc: The number of command line arguments. User should only ever input a single argument
// argv: Stores the command line arguments. The only user argument should be the file to read from.
int main(int argc, char** argv)
{
    // declare a host image object for an 8-bit grayscale image
    npp::ImageCPU_8u_C1 oHostSrc;
    // load gray-scale image from disk
    npp::loadImage("Example.pgn", oHostSrc);
    
    std::cout << oHostSrc.size().nHeight << std::endl;
    std::cout << oHostSrc.size().nWidth << std::endl;
    
    for (int i = 0; i < oHostSrc.size().nHeight; ++i)
    {
        for (int j = 0; j < oHostSrc.size().nWidth; ++j)
        {
            std::cout << oHostSrc.data()[i * oHostSrc.size().nWidth + j] << " ";
        }
        std::cout << std::endl;
    }
    /*// Argc should only have a single argument which is the name of the file to read.
    if (argc != 4)
    {
        std::cout << "Invalid number of arguments. Usage 'gameOfLife.exe # # Example.pgn' where the # are unsigned int for neighborsToGrow, and neighborsToDie respectively." << std::endl;
        
        return -1;
    }
    
    const unsigned int GROW_INDEX = 1;
    const unsigned int DIE_INDEX  = 2;
    const unsigned int PGN_INDEX  = 3;

    unsigned int xSize = 0;
    unsigned int ySize = 0;
    unsigned int neighborsToGrow = std::stoul(argc[GROW_INDEX]);
    unsigned int neighborsToDie = std::stoul(argc[DIE_INDEX]);

    unsigned int * array = (unsigned int*)calloc(xSize * ySize, sizeof(unsigned int));

    executeHost(array, xSize, ySize, neighborsToGrow, neighborsToDie);
    
    
    free(array);*/
    
    return 0;
}
